#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <float.h>
#include "utils.h"

using T = cute::half_t;
using namespace cute;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))

#define OFFSETCOL(row, col, ld) ((col) * (ld) + (row))

template <typename T>
void cpuF16F16Gemm(T *a, T *b, T *c, int M, int N, int K) {

    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += (float)a[OFFSET(m, k, K)] * (float)b[OFFSETCOL(k, n, K)];
            }
            c[OFFSET(m, n, N)] = (T)psum;
        }
    }
}

template <typename T>
float testF16F16GemmMaxError(
    void (*gpuF16F16Gemm) (T *, T *, T *, int, int, int),
    int M, int N, int K) {

    size_t size_a = M * K * sizeof(T);
    size_t size_b = K * N * sizeof(T);
    size_t size_c = M * N * sizeof(T);

    T *h_a, *h_b, *d_a, *d_b;
    T *h_c, *d_c, *h_d_c;

    h_a = (T *)malloc(size_a);
    h_b = (T *)malloc(size_b);
    h_c = (T *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    h_d_c = (T *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = (T)(rand() / float(RAND_MAX));
    for (int i = 0; i < K * N; i++)
        h_b[i] = (T)(rand() / float(RAND_MAX));

    cpuF16F16Gemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

    gpuF16F16Gemm(d_a, d_b, d_c, M, N, K);

    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs((float)h_d_c[i] - (float)h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a); free(h_b); free(h_c); 
    hipFree(d_a); hipFree(d_b); hipFree(d_c); free(h_d_c);

    return max_error;
}

template <typename T, int BM, int BN, int BK, int kStage, typename TiledMMA, 
            typename G2SCopyA, typename G2SCopyB,
            typename SmemLayoutA, typename SmemLayoutB, typename SmemLayoutC,
            typename S2RCopyAtomA, typename S2RCopyAtomB,
            typename R2SCopyAtomC, typename S2GCopyAtomC, typename S2GCopyC>
__global__ void gemm_epilogue_v4(const T *Aptr, const T *Bptr, T *Dptr, int m, int n, int k) {
    // Initilize shared memory
    extern __shared__ T shm_data[];

    T *Ashm = shm_data;
    T *Bshm = shm_data + cute::cosize(SmemLayoutA{});

    // Initilize thread block
    int idx = threadIdx.x;
    int ix = blockIdx.x;
    int iy = blockIdx.y;

    // use Tensor notation to represent device pointer + dimension
    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor D = make_tensor(make_gmem_ptr(Dptr), make_shape(m, n), make_stride(n, Int<1>{}));

    // slice the tensor to small one which is used for current thread block.
    Tensor gA = local_tile(A, make_tile(Int<BM>{}, Int<BK>{}), make_coord(iy, _)); // (BM, BK, num_tile_k)
    Tensor gB = local_tile(B, make_tile(Int<BN>{}, Int<BK>{}), make_coord(ix, _)); // (BN, BK, num_tile_k)
    Tensor gD = local_tile(D, make_tile(Int<BM>{}, Int<BN>{}), make_coord(iy, ix)); // (BM, BN) 

    // shared memory
    auto sA = make_tensor(make_smem_ptr(Ashm),
                            SmemLayoutA{}); // (BM, BK, kStage)
    auto sB = make_tensor(make_smem_ptr(Bshm), SmemLayoutB{}); // (BN, BK, kStage)

    // dispatch TileA/TileB/TileC mma tensor into thread fragment via partition
    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    // auto tCsA = thr_mma.partition_A(sA);                               // (MMA,MMA_M,MMA_K,kStage)
    // auto tCsB = thr_mma.partition_B(sB);                               // (MMA,MMA_N,MMA_K,kStage)
    auto tCgD = thr_mma.partition_C(gD);                                // (MMA, MMA_M, MMA_N)

    auto tCrA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
    auto tCrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrD = thr_mma.partition_fragment_C(gD);           // (MMA, MMA_M, MMA_N)
    clear(tCrD);

    // from global memory to shared memory
    G2SCopyA g2s_tiled_copy_a;
    auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
    auto tAgA_copy = g2s_thr_copy_a.partition_S(gA); // (CPY, CPY_M, CPY_K, k)
    auto tAsA_copy = g2s_thr_copy_a.partition_D(sA); // (CPY, CPY_M, CPY_K, kStage)

    G2SCopyB g2s_tiled_copy_b;
    auto g2s_thr_copy_b = g2s_tiled_copy_b.get_slice(idx);
    auto tBgB_copy = g2s_thr_copy_b.partition_S(gB); // (CPY, CPY_N, CPY_K, k)
    auto tBsB_copy = g2s_thr_copy_b.partition_D(sB); // (CPY, CPY_N, CPY_K, kStage)

    // from shared memory to register, use tiled_mma to generate tiled_copy
    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, tiled_mma);
    auto s2r_thr_copy_a = s2r_tiled_copy_a.get_slice(idx);
    auto tAsA = s2r_thr_copy_a.partition_S(sA);     // (CPY, CPY_M, CPY_K, kStage)
    auto tCrA_view = s2r_thr_copy_a.retile_D(tCrA); // (CPY, CPY_M, CPY_K)

    auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, tiled_mma);
    auto s2r_thr_copy_b = s2r_tiled_copy_b.get_slice(idx);
    auto tBsB = s2r_thr_copy_b.partition_S(sB);     // (CPY, CPY_N, CPY_K, kStage)
    auto tCrB_view = s2r_thr_copy_b.retile_D(tCrB); // (CPY, CPY_N, CPY_K)

    /* PREFETCH */
    // submit kStage - 1 tile
    // gmem -> shm
    int itile_to_read = 0;
    int ismem_read = 0;
    int ismem_write = 0;

#pragma unroll
    for (int istage = 0; istage < kStage - 1; ++istage) {
        cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, istage),
                    tAsA_copy(_, _, _, istage));
        cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, istage),
                    tBsB_copy(_, _, _, istage));
        cp_async_fence();

        ++itile_to_read;
        ++ismem_write;
    }

    // wait one submitted gmem->smem done
    cp_async_wait<kStage - 2>();
    __syncthreads();

    int ik = 0;
    // smem -> reg
    cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik, ismem_read), tCrA_view(_, _, ik));
    cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik, ismem_read), tCrB_view(_, _, ik));

    // loop over k: i. load tile, ii. mma
    int ntile = k / BK;
    #pragma unroll 1
    for (int itile = 0; itile < ntile; ++itile) {
        int nk = size<2>(tCrA);

        #pragma unroll
        for (int ik = 0; ik < nk; ++ik) {
            int ik_next = (ik + 1) % nk;

            if (ik == nk - 1) {
                cp_async_wait<kStage - 2>();
                __syncthreads();

                ismem_read = (ismem_read + 1) % kStage;
            }

            // shm -> reg s[itile][ik + 1] -> r[ik + 1]
            cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik_next, ismem_read),
                    tCrA_view(_, _, ik_next));
            cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik_next, ismem_read),
                    tCrB_view(_, _, ik_next));

            if (ik == 0) {
                if (itile_to_read < ntile) {
                    cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, itile_to_read),
                            tAsA_copy(_, _, _, ismem_write));
                    cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, itile_to_read),
                            tBsB_copy(_, _, _, ismem_write));

                    ++itile_to_read;
                    ismem_write = (ismem_write + 1) % kStage;
                }

                cp_async_fence();
            }

            cute::gemm(tiled_mma, tCrD, tCrA(_, _, ik), tCrB(_, _, ik), tCrD);
        }  // for ik
    }

    // use less shared memory as a scratchpad tile to use large wide instuction
    // Dreg -> shm -> reg -> global
    auto sC = make_tensor(sA(_, _, ismem_read).data(), SmemLayoutC{});

    auto r2s_tiled_copy_c = make_tiled_copy_C(R2SCopyAtomC{}, tiled_mma);
    auto r2s_thr_copy_c = r2s_tiled_copy_c.get_slice(idx);
    auto tCrC_r2s = r2s_thr_copy_c.retile_S(tCrD);   // (CPY, CPY_M, CPY_N)
    auto tCsC_r2s = r2s_thr_copy_c.partition_D(sC);  // (CPY, _1, _1, pipe)

    S2GCopyC s2g_tiled_copy_c;
    auto s2g_thr_copy_c = s2g_tiled_copy_c.get_thread_slice(idx);
    auto tCsC_s2g = s2g_thr_copy_c.partition_S(sC);  // (CPY, _1, _1, pipe)
    auto tCgC_s2g = s2g_thr_copy_c.partition_D(gD);  // (CPY, CPY_M, CPY_N)

    auto tCgC_s2gx = group_modes<1, 3>(tCgC_s2g);  // (CPY_, CPY_MN)
    auto tCrC_r2sx = group_modes<1, 3>(tCrC_r2s);  // (CPY_, CPY_MN)

    int step = size<3>(tCsC_r2s);  // pipe
#pragma unroll
    for (int i = 0; i < size<1>(tCrC_r2sx); i += step) {
        // reg -> shm
#pragma unroll
        for (int j = 0; j < step; ++j) {
            // we add a temp tensor to cope with accumulator and output data type
            // difference
            auto t = make_tensor_like<T>(tCrC_r2sx(_, i + j));
            cute::copy(tCrC_r2sx(_, i + j), t);

            cute::copy(r2s_tiled_copy_c, t, tCsC_r2s(_, 0, 0, j));
        }
        __syncthreads();

    #pragma unroll
        // shm -> global
        for (int j = 0; j < step; ++j) {
            cute::copy(s2g_tiled_copy_c, tCsC_s2g(_, 0, 0, j), tCgC_s2gx(_, i + j));
        }

        __syncthreads();
    }
}

template <typename T>
void gemm_v4(T *a, T *b, T *c, int M, int N, int K) {
    auto BM = Int<128>{};
    auto BN = Int<256>{};
    auto BK = Int<32>{};
    auto KStage = Int<4>{};
    auto kSmemLayoutCBatch = Int<2>{};

    // Define the smem layouts
    using SmemLayoutAtom = decltype(composition(
        Swizzle<3, 3, 3>{},
        make_layout(make_shape(Int<8>{}, Int<BK>{}),
                    make_stride(Int<BK>{}, Int<1>{}))));
    using SmemLayoutA = decltype(tile_to_shape(SmemLayoutAtom{},
                        make_shape(Int<BM>{}, Int<BK>{}, Int<KStage>{})));
    using SmemLayoutB = decltype(tile_to_shape(SmemLayoutAtom{},
                        make_shape(Int<BN>{}, Int<BK>{}, Int<KStage>{})));                    // (m,n) -> smem_idx
    
    // mma
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int kMmaEURepeatM = 2;
    static constexpr int kMmaEURepeatN = 2;
    static constexpr int kMmaEURepeatK = 1;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
    static constexpr int kMmaPN = 2 * kMmaEURepeatN * get<1>(mma_atom_shape{});
    static constexpr int kMmaPK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});
    using MMA_EU_RepeatT = decltype(make_layout(make_shape(
        Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));
    
    // copy from global memory to shared memory
    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>;
    using G2SCopyA =
        decltype(make_tiled_copy(g2s_copy_atom{},
                                 make_layout(make_shape(Int<32>{}, Int<4>{}), // Thr layout 32x4 k-major
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{})))); // Val layout 1x8
    using G2SCopyB = G2SCopyA;

    // copy from shared memory to register
    // use mma tiled ,so no tiled here
    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
    using S2RCopyAtomA = s2r_copy_atom;
    using S2RCopyAtomB = s2r_copy_atom;

    // epilogue: register to global via shared memory
    using SmemLayoutAtomC = decltype(composition(
    Swizzle<3, 3, 3>{}, make_layout(make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}),
                                    make_stride(Int<kMmaPN>{}, Int<1>{}))));
    using SmemLayoutC = decltype(tile_to_shape(
        SmemLayoutAtomC{},
        make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}, Int<kSmemLayoutCBatch>{})));

    static_assert(size<0>(SmemLayoutA{}) * size<1>(SmemLayoutA{}) >=
                    size(SmemLayoutC{}),
                "C shared memory request is large than A's one pipe");

    using R2SCopyAtomC = Copy_Atom<UniversalCopy<int>, T>;

    using S2GCopyAtomC = Copy_Atom<UniversalCopy<cute::uint128_t>, T>;
    using S2GCopyC =
        decltype(make_tiled_copy(S2GCopyAtomC{},
                                make_layout(make_shape(Int<32>{}, Int<4>{}),
                                            make_stride(Int<4>{}, Int<1>{})),
                                make_layout(make_shape(Int<1>{}, Int<8>{}))));

    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    dim3 block(size(MMA{}));
    dim3 grid(BX, BY);

    // C_shm is shared with A_shm and B_shm
    static constexpr int shm_size_AB =
        cute::cosize(SmemLayoutA{}) + cute::cosize(SmemLayoutB{});
    static constexpr int shm_size_C = cute::cosize(SmemLayoutC{});
    static constexpr int kShmSize =
        cute::max(shm_size_AB, shm_size_C) * sizeof(T);

    int shm_size = kShmSize;

    hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_epilogue_v4<T), BM, BN, BK, KStage, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, SmemLayoutC, S2RCopyAtomA, S2RCopyAtomB, R2SCopyAtomC, S2GCopyAtomC, S2GCopyC>,
                         hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    
    gemm_epilogue_v4<T, BM, BN, BK, KStage, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, SmemLayoutC, S2RCopyAtomA, S2RCopyAtomB, R2SCopyAtomC, S2GCopyAtomC, S2GCopyC>
               <<<grid, block, shm_size>>>(a, b, c, M, N, K);
}


template <typename T>
float testF16F16GemmPerformance(
    void (*gpuF16F16Gemm) (T *, T *, T *, int, int, int),
    int M, int N, int K, int repeat) {

    size_t size_a = M * K * sizeof(T);
    size_t size_b = K * N * sizeof(T);
    size_t size_c = M * N * sizeof(T);

    T *d_a, *d_b;
    T *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++) {
        gpuF16F16Gemm(d_a, d_b, d_c, M, N, K);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return sec;
}


int main() {
    const int test_num = 64;
    int M_list[test_num];
    int N_list[test_num];
    int K_list[test_num];

    for (int i = 0; i < test_num; i++) {
        M_list[i] = (i + 1) * 256;
        N_list[i] = (i + 1) * 256;
        K_list[i] = (i + 1) * 256;
    }

    const int outer_repeat = 10, inner_repeat = 1;

    printf("\nalgo = Cute_HGEMM_V3\n");

    const int M = 1024, N = 1024, K = 1024;
    float max_error = testF16F16GemmMaxError<T>(
        gemm_v4, M, N, K);
    printf("Max Error = %f\n", max_error);

    for (int j = 0; j < test_num; j++) {
        int M = M_list[j], N = N_list[j], K = K_list[j];

        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int k = 0; k < outer_repeat; k++) {
            double this_sec = testF16F16GemmPerformance<T>(
                gemm_v4, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

        printf("M N K = %6d %6d %6d, ", M, N, K);
        printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
        printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
    }

    return 0;
}
