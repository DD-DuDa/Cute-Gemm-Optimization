#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"

#define PRINT(name, content) \
    print(name);             \
    print(" : ");            \
    print(content);          \
    print("\n");

#define PRINTTENSOR(name, content) \
    print(name);                   \
    print(" : ");                  \
    print_tensor(content);         \
    print("\n");

/// Simple copy kernel.
//
// Uses local_partition() to partition a tile among threads arranged as (THR_M, THR_N).
template <class TensorS, class TensorD, class ThreadLayout>
__global__ void copy_kernel(TensorS S, TensorD D, ThreadLayout)
{
    using namespace cute;

    // Slice the tiled tensors
    Tensor tile_S = S(make_coord(_,_), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)
    Tensor tile_D = D(make_coord(_,_), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)

    // Construct a partitioning of the tile among threads with the given thread arrangement.

    // Concept:                         Tensor  ThrLayout       ThrIndex
    Tensor thr_tile_S = local_partition(tile_S, ThreadLayout{}, threadIdx.x);  // (ThrValM, ThrValN)
    Tensor thr_tile_D = local_partition(tile_D, ThreadLayout{}, threadIdx.x);  // (ThrValM, ThrValN)

    // Construct a register-backed Tensor with the same shape as each thread's partition
    // Use make_tensor to try to match the layout of thr_tile_S
    Tensor fragment = make_tensor_like(thr_tile_S);               // (ThrValM, ThrValN)

    // Copy from GMEM to RMEM and from RMEM to GMEM
    copy(thr_tile_S, fragment);
    copy(fragment, thr_tile_D);
}

/// Vectorized copy kernel.
///
/// Uses `make_tiled_copy()` to perform a copy using vector instructions. This operation
/// has the precondition that pointers are aligned to the vector size.
///
template <class TensorS, class TensorD, class ThreadLayout, class VecLayout>
__global__ void copy_kernel_vectorized(TensorS S, TensorD D, ThreadLayout, VecLayout)
{
    using namespace cute;
    using Element = typename TensorS::value_type;

    // Slice the tensors to obtain a view into each tile.
    Tensor tile_S = S(make_coord(_, _), blockIdx.x, blockIdx.y);  // (BlockShape_M, BlockShape_N)
    Tensor tile_D = D(make_coord(_, _), blockIdx.x, blockIdx.y);  // (BlockShape_M, BlockShape_N)

    // Define `AccessType` which controls the size of the actual memory access.
    using AccessType = cutlass::AlignedArray<Element, size(VecLayout{})>;

    // A copy atom corresponds to one hardware memory access.
    using Atom = Copy_Atom<UniversalCopy<AccessType>, Element>;

    // Construct tiled copy, a tiling of copy atoms.
    //
    // Note, this assumes the vector and thread layouts are aligned with contigous data
    // in GMEM. Alternative thread layouts are possible but may result in uncoalesced
    // reads. Alternative vector layouts are also possible, though incompatible layouts
    // will result in compile time errors.
    auto tiled_copy =
        make_tiled_copy(
        Atom{},                       // access size
        ThreadLayout{},               // thread layout
        VecLayout{});                 // vector layout (e.g. 4x1)

    // Construct a Tensor corresponding to each thread's slice.
    auto thr_copy = tiled_copy.get_thread_slice(threadIdx.x);

    Tensor thr_tile_S = thr_copy.partition_S(tile_S);             // (CopyOp, CopyM, CopyN)
    Tensor thr_tile_D = thr_copy.partition_D(tile_D);             // (CopyOp, CopyM, CopyN)

    if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        PRINT("tile_S", tile_S.layout())
        PRINT("tile_D", tile_D.layout())
        PRINT("thr_tile_S", thr_tile_S.layout())
        PRINT("thr_tile_D", thr_tile_D.layout())
    }

    // Construct a register-backed Tensor with the same shape as each thread's partition
    // Use make_fragment because the first mode is the instruction-local mode
    Tensor fragment = make_fragment_like(thr_tile_D);             // (CopyOp, CopyM, CopyN)

    // Copy from GMEM to RMEM and from RMEM to GMEM
    copy(tiled_copy, thr_tile_S, fragment);
    copy(tiled_copy, fragment, thr_tile_D);
}

/// Main function
int main(int argc, char** argv)
{
    using namespace cute;
    using Element = float;

    // Define a tensor shape with dynamic extents (m, n)
    auto tensor_shape = make_shape(256, 512);

    //
    // Allocate and initialize
    //

    thrust::host_vector<Element> h_S(size(tensor_shape));
    thrust::host_vector<Element> h_D(size(tensor_shape));

    for (size_t i = 0; i < h_S.size(); ++i) {
        h_S[i] = static_cast<Element>(i);
        h_D[i] = Element{};
    }

    thrust::device_vector<Element> d_S = h_S;
    thrust::device_vector<Element> d_D = h_D;

    //
    // Make tensors
    //

    Tensor tensor_S = make_tensor(make_gmem_ptr(thrust::raw_pointer_cast(d_S.data())), make_layout(tensor_shape));
    Tensor tensor_D = make_tensor(make_gmem_ptr(thrust::raw_pointer_cast(d_D.data())), make_layout(tensor_shape));


    //
    // Tile tensors
    //

    // Define a statically sized block (M, N).
    // Note, by convention, capital letters are used to represent static modes.
    auto block_shape = make_shape(Int<128>{}, Int<64>{});

    // Tile the tensor (m, n) ==> ((M, N), m', n') where (M, N) is the static tile
    // shape, and modes (m', n') correspond to the number of tiles.
    //
    // These will be used to determine the CUDA kernel grid dimensions.
    Tensor tiled_tensor_S = tiled_divide(tensor_S, block_shape);      // ((M, N), m', n') -> ((_128,_64),2,8)
    Tensor tiled_tensor_D = tiled_divide(tensor_D, block_shape);      // ((M, N), m', n') -> ((_128,_64),2,8)((_128,_64),2,8)

    PRINT("tiled_tensor_S", tiled_tensor_S.shape())

    // Thread arrangement
    Layout thr_layout = make_layout(make_shape(Int<32>{}, Int<8>{}));

    // Vector dimensions
    Layout vec_layout = make_layout(make_shape(Int<4>{}, Int<1>{}));

    //
    // Determine grid and block dimensions
    //

    dim3 gridDim (size<1>(tiled_tensor_D), size<2>(tiled_tensor_D));   // Grid shape corresponds to modes m' and n'
    dim3 blockDim(size(thr_layout));

    //
    // Launch the kernel
    //
    copy_kernel<<< gridDim, blockDim >>>(
        tiled_tensor_S,
        tiled_tensor_D,
        thr_layout);

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime error: " << hipGetErrorString(result) << std::endl;
        return -1;
    }

    //
    // Verify
    //

    h_D = d_D;

    int32_t errors = 0;
    int32_t const kErrorLimit = 10;

    for (size_t i = 0; i < h_D.size(); ++i) {
        if (h_S[i] != h_D[i]) {
        std::cerr << "Error. S[" << i << "]: " << h_S[i] << ",   D[" << i << "]: " << h_D[i] << std::endl;

        if (++errors >= kErrorLimit) {
            std::cerr << "Aborting on " << kErrorLimit << "nth error." << std::endl;
            return -1;
        }
        }
    }

    std::cout << "Success." << std::endl;

    return 0;
}