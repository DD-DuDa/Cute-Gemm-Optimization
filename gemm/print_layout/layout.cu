#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <float.h>

using T = cute::half_t;
using namespace cute;

int main() {
    TiledMMA mma = make_tiled_mma(SM80_16x8x16_F32F16F16F32_TN{},
                                  Layout<Shape<_4, _1, _1>>{},
                                  Tile<Int<16 * 4>, _16, _16>{});           // Tiler
    print_latex(mma);

    return 0;
}