#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <float.h>
#include "utils.h"

using T = cute::half_t;
using namespace cute;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))

#define OFFSETCOL(row, col, ld) ((col) * (ld) + (row))

template <typename T>
void cpuF16F16Gemm(T *a, T *b, T *c, int M, int N, int K) {

    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += (float)a[OFFSET(m, k, K)] * (float)b[OFFSETCOL(k, n, K)];
            }
            c[OFFSET(m, n, N)] = (T)psum;
        }
    }
}

template <typename T, int BM, int BN, int BK, typename TiledMMA, 
            typename G2SCopyA, typename G2SCopyB,
            typename SmemLayoutA, typename SmemLayoutB, 
            typename S2RCopyAtomA, typename S2RCopyAtomB>
__global__ void gemm_shm_v2(const T *Aptr, const T *Bptr, T *Dptr, int m, int n, int k) {
    // Initilize shared memory
    extern __shared__ T shm_data[];

    T *Ashm = shm_data;
    T *Bshm = shm_data + cute::cosize(SmemLayoutA{});

    // Initilize thread block
    int idx = threadIdx.x;
    int ix = blockIdx.x;
    int iy = blockIdx.y;

    
    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor D = make_tensor(make_gmem_ptr(Dptr), make_shape(m, n), make_stride(n, Int<1>{}));

    // Global Memory
    Tensor gA = local_tile(A, make_tile(Int<BM>{}, Int<BK>{}), make_coord(iy, _)); // (BM, BK, num_tile_k)
    Tensor gB = local_tile(B, make_tile(Int<BN>{}, Int<BK>{}), make_coord(ix, _)); // (BN, BK, num_tile_k)
    Tensor gD = local_tile(D, make_tile(Int<BM>{}, Int<BN>{}), make_coord(iy, ix)); // (BM, BN) 

    // Compute tile residues for predication
    auto m_max_coord = m - size<0>(gA) * iy;        // M - BM * m_coord
    auto n_max_coord = n - size<0>(gB) * ix;        // N - BN * n_coord
    auto k_residue = k - size<1>(gA) * size<2>(gA); // K - BK * k_coord_max
    auto residue_mnk = make_tuple(m_max_coord, n_max_coord, k_residue);

    // Shift tensor so residue_k is at origin (Can't read any k_coord < residue_k)
    // This aligns the tensor with BLK_K for all but the 0th k_tile
    gA = cute::domain_offset(make_coord(0, get<2>(residue_mnk), 0), gA);
    gB = cute::domain_offset(make_coord(0, get<2>(residue_mnk), 0), gB);

    // shared memory
    auto sA = make_tensor(make_smem_ptr(Ashm),
                            SmemLayoutA{}); // (BM, BK)
    auto sB = make_tensor(make_smem_ptr(Bshm), SmemLayoutB{}); // (BN, BK)

    // register, use tiled_mma to partition register A/B/C
    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    auto tCgD = thr_mma.partition_C(gD); // (MMA, MMA_M, MMA_N)

    auto tCrA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
    auto tCrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrD = thr_mma.partition_fragment_C(gD);           // (MMA, MMA_M, MMA_N)
    clear(tCrD);

    // from global memory to shared memory
    G2SCopyA g2s_tiled_copy_a;
    auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
    auto tAgA_copy = g2s_thr_copy_a.partition_S(gA); // (CPY, CPY_M, CPY_K, k)
    auto tAsA_copy = g2s_thr_copy_a.partition_D(sA); // (CPY, CPY_M, CPY_K)

    G2SCopyB g2s_tiled_copy_b;
    auto g2s_thr_copy_b = g2s_tiled_copy_b.get_slice(idx);
    auto tBgB_copy = g2s_thr_copy_b.partition_S(gB); // (CPY, CPY_N, CPY_K, k)
    auto tBsB_copy = g2s_thr_copy_b.partition_D(sB); // (CPY, CPY_N, CPY_K)

    // from shared memory to register, use tiled_mma to generate tiled_copy
    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, tiled_mma);
    auto s2r_thr_copy_a = s2r_tiled_copy_a.get_slice(idx);
    auto tAsA = s2r_thr_copy_a.partition_S(sA);     // (CPY, CPY_M, CPY_K)
    auto tCrA_view = s2r_thr_copy_a.retile_D(tCrA); // (CPY, CPY_M, CPY_K)

    auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, tiled_mma);
    auto s2r_thr_copy_b = s2r_tiled_copy_b.get_slice(idx);
    auto tBsB = s2r_thr_copy_b.partition_S(sB);     // (CPY, CPY_N, CPY_K)
    auto tCrB_view = s2r_thr_copy_b.retile_D(tCrB); // (CPY, CPY_N, CPY_K)

    // Construct identity layout for sA and sB
    Tensor cA = make_identity_tensor(make_shape(size<0>(sA), size<1>(sA)));    // (BLK_M,BLK_K) -> (blk_m,blk_k)
    Tensor cB = make_identity_tensor(make_shape(size<0>(sB), size<1>(sB)));    // (BLK_N,BLK_K) -> (blk_n,blk_k)

    // Repeat the partitioning with identity layouts
    Tensor tAcA = g2s_thr_copy_a.partition_S(cA);                             // (ACPY,ACPY_M,ACPY_K) -> (blk_m,blk_k)
    Tensor tBcB = g2s_thr_copy_b.partition_S(cB);                             // (BCPY,BCPY_N,BCPY_K) -> (blk_n,blk_k)

    // Clear the smem tiles to account for predicated off loads
    clear(tAsA_copy);
    clear(tBsB_copy);

    Tensor tAgAk = tAgA_copy(_,_,_,0);
    CUTLASS_PRAGMA_UNROLL
    for (int k = 0; k < size<2>(tAsA_copy); ++k) {
        if (get<1>(tAcA(0,0,k)) >= -get<2>(residue_mnk)) {      // blk_k coord < residue_k (gA shifted)
            cute::copy(g2s_tiled_copy_a, tAgAk(_,_,k), tAsA_copy(_,_,k));
        }
    }
    Tensor tBgBk = tBgB_copy(_,_,_,0);
    CUTLASS_PRAGMA_UNROLL
    for (int k = 0; k < size<2>(tBsB_copy); ++k) {
        if (get<1>(tBcB(0,0,k)) >= -get<2>(residue_mnk)) {      // blk_k coord < residue_k (gA shifted)
            cute::copy(g2s_tiled_copy_b, tBgBk(_,_,k), tBsB_copy(_,_,k));
        }
    }
    cp_async_fence();
    cp_async_wait<0>();
    __syncthreads();

    if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    {

        PRINT("sA", sA.shape())   
        PRINT("tAsA", tAsA.shape())   
        PRINT("tCrA_view", tCrA_view.shape())    
        // printf("size<2>(tAsA_copy):%d \n", size<2>(tAsA_copy));
        PRINT("sB", sB.shape())
        PRINT("tBsB", tBsB.shape())
        PRINT("tCrB_view", tCrB_view.shape()) 
    }

  // loop over k: i. load tile, ii. mma
  int ntile = k / BK;
#pragma unroll 1
  for (int itile = 0; itile < ntile; ++itile)
  {
    if (itile >= 1) {
        // copy  (CPY, CPY_M, CPY_K) , async
        cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, itile),
            tAsA_copy(_, _, _));
        cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, itile),
            tBsB_copy(_, _, _));
        cp_async_fence();

        cp_async_wait<0>();
        __syncthreads();
    }
    
    __syncthreads();
    int nk = size<2>(tCrA);
#pragma unroll
    for (int ik = 0; ik < nk; ++ik)
    {
      // copy  (CPY, CPY_M), sync
      cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik),
                 tCrA_view(_, _, ik));
      // copy  (CPY, CPY_N)
      cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik),
                 tCrB_view(_, _, ik));
      // (MMA, MMA_M) x (MMA, MMA_N) => (MMA, MMA_M, MMA_N)
      cute::gemm(tiled_mma, tCrD, tCrA(_, _, ik), tCrB(_, _, ik), tCrD);
    } // for ik
    
  } // itile

  // register to global memory
  cute::copy(tCrD, tCgD);
}

template <typename T>
void gemm_v2(T *a, T *b, T *c, int M, int N, int K) {

    auto BM = Int<128>{};
    auto BN = Int<256>{};
    auto BK = Int< 32>{};
    // Define the smem layouts
    using SmemLayoutAtom = decltype(composition(
        Swizzle<3, 3, 3>{},
        make_layout(make_shape(Int<8>{}, Int<BK>{}),
                    make_stride(Int<BK>{}, Int<1>{}))));
    using SmemLayoutA = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BM>{}, Int<BK>{})));
    using SmemLayoutB = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BN>{}, Int<BK>{})));                    // (m,n) -> smem_idx

    // mma
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int kMmaEURepeatM = 2;
    static constexpr int kMmaEURepeatN = 2;
    static constexpr int kMmaEURepeatK = 1;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
    static constexpr int kMmaPN = 2 * kMmaEURepeatN * get<1>(mma_atom_shape{});
    static constexpr int kMmaPK = 2 * kMmaEURepeatK * get<2>(mma_atom_shape{});
    using MMA_EU_RepeatT = decltype(make_layout(make_shape(
        Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
  
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));

    // copy from global memory to shared memory
    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>;
    using G2SCopyA =
        decltype(make_tiled_copy(g2s_copy_atom{},
                                 make_layout(make_shape(Int<32>{}, Int<4>{}), // Thr layout 32x4 k-major
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{})))); // Val layout 1x8
    using G2SCopyB = G2SCopyA;

    // copy from shared memory to register
    // use mma tiled ,so no tiled here
    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
    using S2RCopyAtomA = s2r_copy_atom;
    using S2RCopyAtomB = s2r_copy_atom;

    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    dim3 block(size(MMA{}));
    dim3 grid(BX, BY);

    // C_shm is shared with A_shm and B_shm
    static constexpr int shm_size_AB =
        cute::cosize(SmemLayoutA{}) + cute::cosize(SmemLayoutB{});
    static constexpr int kShmSize =
        shm_size_AB * sizeof(T);

    int shm_size = kShmSize;

    hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_shm_v2<T), BM, BN, BK, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, S2RCopyAtomA, S2RCopyAtomB>,
                         hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    
    gemm_shm_v2<T, BM, BN, BK, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, S2RCopyAtomA, S2RCopyAtomB>
               <<<grid, block, shm_size>>>(a, b, c, M, N, K);
}


template <typename T>
float testF16F16GemmMaxError(
    void (*gpuF16F16Gemm) (T *, T *, T *, int, int, int),
    int M, int N, int K) {

    size_t size_a = M * K * sizeof(T);
    size_t size_b = K * N * sizeof(T);
    size_t size_c = M * N * sizeof(T);

    T *h_a, *h_b, *d_a, *d_b;
    T *h_c, *d_c, *h_d_c;

    h_a = (T *)malloc(size_a);
    h_b = (T *)malloc(size_b);
    h_c = (T *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    h_d_c = (T *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = (T)(rand() / float(RAND_MAX));
    for (int i = 0; i < K * N; i++)
        h_b[i] = (T)(rand() / float(RAND_MAX));

    cpuF16F16Gemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

    gpuF16F16Gemm(d_a, d_b, d_c, M, N, K);

    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        printf("\n h_c : %f \n", (float)h_c[i]);
        printf("\n h_d_c : %f \n", (float)h_d_c[i]);
    }
    
    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs((float)h_d_c[i] - (float)h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a); free(h_b); free(h_c); 
    hipFree(d_a); hipFree(d_b); hipFree(d_c); free(h_d_c);

    return max_error;
}

template <typename T>
float testF16F16GemmPerformance(
    void (*gpuF16F16Gemm) (T *, T *, T *, int, int, int),
    int M, int N, int K, int repeat) {

    size_t size_a = M * K * sizeof(T);
    size_t size_b = K * N * sizeof(T);
    size_t size_c = M * N * sizeof(T);

    T *d_a, *d_b;
    T *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++) {
        gpuF16F16Gemm(d_a, d_b, d_c, M, N, K);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return sec;
}


int main() {
    const int test_num = 64;
    int M_list[test_num];
    int N_list[test_num];
    int K_list[test_num];

    for (int i = 0; i < test_num; i++) {
        M_list[i] = (i + 1) * 256;
        N_list[i] = (i + 1) * 256;
        K_list[i] = (i + 1) * 256;
    }

    const int outer_repeat = 10, inner_repeat = 1;

    printf("\nalgo = Cute_HGEMM_V2\n");

    const int M = 1024, N = 1024, K = 1025;
    float max_error = testF16F16GemmMaxError<T>(
        gemm_v2, M, N, K);
    printf("Max Error = %f\n", max_error);

    // double this_sec = testF16F16GemmPerformance<T>(
    //     gemm_v2, 8192, 8192, 8192, inner_repeat);
    // for (int j = 0; j < test_num; j++) {
    //     int M = M_list[j], N = N_list[j], K = K_list[j];

    //     double max_sec = 0.0;
    //     double min_sec = DBL_MAX;
    //     double total_sec = 0.0;

    //     for (int k = 0; k < outer_repeat; k++) {
    //         double this_sec = testF16F16GemmPerformance<T>(
    //             gemm_v2, M, N, K, inner_repeat);
    //         max_sec = max(max_sec, this_sec);
    //         min_sec = min(min_sec, this_sec);
    //         total_sec += this_sec;
    //     }

    //     double avg_sec = total_sec / outer_repeat;
    //     double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

    //     printf("M N K = %6d %6d %6d, ", M, N, K);
    //     printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
    //     printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
    // }

    return 0;
}