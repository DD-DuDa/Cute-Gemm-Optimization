#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <float.h>
#include "utils.h"

using T = cute::half_t;
using namespace cute;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))

#define OFFSETCOL(row, col, ld) ((col) * (ld) + (row))

template <typename T>
void cpuF16F16Gemm(T *a, T *b, T *c, int M, int N, int K) {

    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += (float)a[OFFSET(m, k, K)] * (float)b[OFFSETCOL(k, n, K)];
            }
            c[OFFSET(m, n, N)] = (T)psum;
        }
    }
}

template <typename T, int BM, int BN, int BK, typename TiledMMA, 
            typename G2SCopyA, typename G2SCopyB,
            typename SmemLayoutA, typename SmemLayoutB, typename SmemLayoutC,
            typename S2RCopyAtomA, typename S2RCopyAtomB,
            typename R2SCopyAtomC, typename S2GCopyAtomC, typename S2GCopyC>
__global__ void gemm_shm_v5(const T *Aptr, const T *Bptr, T *Dptr, int m, int n, int k) {
    // Initilize shared memory
    extern __shared__ T shm_data[];

    T *Ashm = shm_data;
    T *Bshm = shm_data + cute::cosize(SmemLayoutA{});

    // Initilize thread block
    int idx = threadIdx.x;
    int ix = blockIdx.x;
    int iy = blockIdx.y;

    
    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor D = make_tensor(make_gmem_ptr(Dptr), make_shape(m, n), make_stride(n, Int<1>{}));

    // Global Memory
    Tensor gA = local_tile(A, make_tile(Int<BM>{}, Int<BK>{}), make_coord(iy, _)); // (BM, BK, num_tile_k)
    Tensor gB = local_tile(B, make_tile(Int<BN>{}, Int<BK>{}), make_coord(ix, _)); // (BN, BK, num_tile_k)
    Tensor gD = local_tile(D, make_tile(Int<BM>{}, Int<BN>{}), make_coord(iy, ix)); // (BM, BN) 

    // shared memory
    auto sA = make_tensor(make_smem_ptr(Ashm),
                            SmemLayoutA{}); // (BM, BK)
    auto sB = make_tensor(make_smem_ptr(Bshm), SmemLayoutB{}); // (BN, BK)

    // register, use tiled_mma to partition register A/B/C
    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    auto tCgD = thr_mma.partition_C(gD); // (MMA, MMA_M, MMA_N)

    auto tCrA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
    auto tCrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrD = thr_mma.partition_fragment_C(gD);           // (MMA, MMA_M, MMA_N)
    clear(tCrD);

    // from global memory to shared memory
    G2SCopyA g2s_tiled_copy_a;
    auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
    auto tAgA_copy = g2s_thr_copy_a.partition_S(gA); // (CPY, CPY_M, CPY_K, k)
    auto tAsA_copy = g2s_thr_copy_a.partition_D(sA); // (CPY, CPY_M, CPY_K)

    G2SCopyB g2s_tiled_copy_b;
    auto g2s_thr_copy_b = g2s_tiled_copy_b.get_slice(idx);
    auto tBgB_copy = g2s_thr_copy_b.partition_S(gB); // (CPY, CPY_N, CPY_K, k)
    auto tBsB_copy = g2s_thr_copy_b.partition_D(sB); // (CPY, CPY_N, CPY_K)

    // from shared memory to register, use tiled_mma to generate tiled_copy
    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, tiled_mma);
    auto s2r_thr_copy_a = s2r_tiled_copy_a.get_slice(idx);
    auto tAsA = s2r_thr_copy_a.partition_S(sA);     // (CPY, CPY_M, CPY_K)
    auto tCrA_view = s2r_thr_copy_a.retile_D(tCrA); // (CPY, CPY_M, CPY_K)

    auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, tiled_mma);
    auto s2r_thr_copy_b = s2r_tiled_copy_b.get_slice(idx);
    auto tBsB = s2r_thr_copy_b.partition_S(sB);     // (CPY, CPY_N, CPY_K)
    auto tCrB_view = s2r_thr_copy_b.retile_D(tCrB); // (CPY, CPY_N, CPY_K)


    // if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     PRINT("tCrA", tCrA.shape())    
    //     PRINT("tCrB", tCrB.shape())   

    //     PRINT("tAgA_copy", tAgA_copy.shape())     
    //     PRINT("tAsA_copy", tAsA_copy.shape())
    //     // print(layout<0>(tAgA));
    //     // PRINT("tArA", tArA.shape()) 
    //     PRINT("tBgB_copy", tBgB_copy.shape())     
    //     PRINT("tBsB_copy", tBsB_copy.shape())

    //     PRINT("tAsA", tAsA.shape())     
    //     PRINT("tCrA_view", tCrA_view.shape()) 
    //     // print(layout<0>(tBgB));
    //     // PRINT("tBrB", tBrB.shape()) 

    //     PRINT("tBsB", tBsB.shape())     
    //     PRINT("tCrB_view", tCrB_view.shape()) 
    // }

    // loop over k: i. load tile, ii. mma
    int ntile = k / BK;
    #pragma unroll 1
    for (int itile = 0; itile < ntile; ++itile)
    {
        // copy  (CPY, CPY_M, CPY_K) , async
        cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, itile),
                tAsA_copy(_, _, _));
        cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, itile),
                tBsB_copy(_, _, _));
        cp_async_fence();

        cp_async_wait<0>();
        __syncthreads();

        int nk = size<2>(tCrA);
    #pragma unroll
        for (int ik = 0; ik < nk; ++ik)
        {
            // copy  (CPY, CPY_M), sync
            cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik),
                        tCrA_view(_, _, ik));
            // copy  (CPY, CPY_N)
            cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik),
                        tCrB_view(_, _, ik));
            // (MMA, MMA_M) x (MMA, MMA_N) => (MMA, MMA_M, MMA_N)
            cute::gemm(tiled_mma, tCrD, tCrA(_, _, ik), tCrB(_, _, ik), tCrD);
        } // for ik
    } // itile

  
    // use less shared memory as a scratchpad tile to use large wide instuction
    // Dreg -> shm -> reg -> global
    auto sC = make_tensor(sA(_, _).data(), SmemLayoutC{});

    auto r2s_tiled_copy_c = make_tiled_copy_C(R2SCopyAtomC{}, tiled_mma);
    auto r2s_thr_copy_c = r2s_tiled_copy_c.get_slice(idx);
    auto tCrC_r2s = r2s_thr_copy_c.retile_S(tCrD);   // (CPY, CPY_M, CPY_N) ((_2,(_2,_2)),_4,_8)
    auto tCsC_r2s = r2s_thr_copy_c.partition_D(sC);  // (CPY, _1, _1, pipe) ((_2,(_2,_2)),_1,_1,_2)

    S2GCopyC s2g_tiled_copy_c;
    auto s2g_thr_copy_c = s2g_tiled_copy_c.get_thread_slice(idx);
    auto tCsC_s2g = s2g_thr_copy_c.partition_S(sC);  // (CPY, _1, _1, pipe) ((_2,(_2,_2)),_1,_1,_2)
    auto tCgC_s2g = s2g_thr_copy_c.partition_D(gD);  // (CPY, CPY_M, CPY_N) ((_8,_1),_4,_8)

    
    auto tCrC_r2sx = group_modes<1, 3>(tCrC_r2s);  // (CPY_, CPY_MN) ((_2,(_2,_2)),(_4,_8))
    auto tCgC_s2gx = group_modes<1, 3>(tCgC_s2g);  // (CPY_, CPY_MN) ((_8,_1),(_4,_8))

    // if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
    //     PRINT("tCrC_r2s", tCrC_r2s.shape()) 
    //     PRINT("tCsC_r2s", tCsC_r2s.shape()) 

    //     // PRINT("tCsC_s2g", tCsC_s2g.shape()) 
    //     // PRINT("tCgC_s2g", tCgC_s2g.shape()) 

    //     // PRINT("tCgC_s2gx", tCgC_s2gx.shape()) 
    //     PRINT("tCrC_r2sx", tCrC_r2sx.shape()) 
    //     PRINT("size<1>(tCrC_r2sx)", size<1>(tCrC_r2sx))
    // }

    int step = size<3>(tCsC_r2s);  // pipe
#pragma unroll
    for (int i = 0; i < size<1>(tCrC_r2sx); i += step) {
        // reg -> shm
#pragma unroll
        for (int j = 0; j < step; ++j) {
            // we add a temp tensor to cope with accumulator and output data type
            // difference
            auto t = make_tensor_like<T>(tCrC_r2sx(_, i + j));
            cute::copy(tCrC_r2sx(_, i + j), t);

            cute::copy(r2s_tiled_copy_c, t, tCsC_r2s(_, 0, 0, j));
        }
        __syncthreads();

    #pragma unroll
        // shm -> global
        for (int j = 0; j < step; ++j) {
            cute::copy(s2g_tiled_copy_c, tCsC_s2g(_, 0, 0, j), tCgC_s2gx(_, i + j));
        }

        __syncthreads();
    }
}

template <typename T>
void gemm_v3(T *a, T *b, T *c, int M, int N, int K) {
    auto BM = Int<128>{};
    auto BN = Int<256>{};
    auto BK = Int< 32>{};
    auto kSmemLayoutCBatch = Int<2>{};
    // Define the smem layouts
    using SmemLayoutAtom = decltype(composition(
        Swizzle<3, 3, 3>{},
        make_layout(make_shape(Int<8>{}, Int<BK>{}),
                    make_stride(Int<BK>{}, Int<1>{}))));
    using SmemLayoutA = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BM>{}, Int<BK>{})));
    using SmemLayoutB = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BN>{}, Int<BK>{})));                    // (m,n) -> smem_idx

    // mma
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int kMmaEURepeatM = 2;
    static constexpr int kMmaEURepeatN = 2;
    static constexpr int kMmaEURepeatK = 1;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
    static constexpr int kMmaPN = 2 * kMmaEURepeatN * get<1>(mma_atom_shape{});
    static constexpr int kMmaPK = 2 * kMmaEURepeatK * get<2>(mma_atom_shape{});
    using MMA_EU_RepeatT = decltype(make_layout(make_shape(
        Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));

    // copy from global memory to shared memory
    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>;
    using G2SCopyA =
        decltype(make_tiled_copy(g2s_copy_atom{},
                                 make_layout(make_shape(Int<32>{}, Int<4>{}), // Thr layout 32x4 k-major
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{})))); // Val layout 1x8
    using G2SCopyB = G2SCopyA;

    // copy from shared memory to register
    // use mma tiled ,so no tiled here
    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
    using S2RCopyAtomA = s2r_copy_atom;
    using S2RCopyAtomB = s2r_copy_atom;

    // epilogue: register to global via shared memory
    using SmemLayoutAtomC = decltype(composition(
        Swizzle<3, 3, 3>{}, make_layout(make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}),
                                        make_stride(Int<kMmaPN>{}, Int<1>{}))));
    using SmemLayoutC = decltype(tile_to_shape(
        SmemLayoutAtomC{},
        make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}, Int<kSmemLayoutCBatch>{})));

    static_assert(size<0>(SmemLayoutA{}) * size<1>(SmemLayoutA{}) >=
                    size(SmemLayoutC{}),
                "C shared memory request is large than A's one pipe");

    using R2SCopyAtomC = Copy_Atom<UniversalCopy<int>, T>;
    using S2GCopyAtomC = Copy_Atom<UniversalCopy<cute::uint128_t>, T>;
    using S2GCopyC =
        decltype(make_tiled_copy(S2GCopyAtomC{},
                                make_layout(make_shape(Int<32>{}, Int<4>{}),
                                            make_stride(Int<4>{}, Int<1>{})),
                                make_layout(make_shape(Int<1>{}, Int<8>{}))));

    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    dim3 block(size(MMA{}));
    dim3 grid(BX, BY);

    // C_shm is shared with A_shm and B_shm
    static constexpr int shm_size_AB =
        cute::cosize(SmemLayoutA{}) + cute::cosize(SmemLayoutB{});
    static constexpr int shm_size_C = cute::cosize(SmemLayoutC{});
    static constexpr int kShmSize =
        cute::max(shm_size_AB, shm_size_C) * sizeof(T);

    int shm_size = kShmSize;

    hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_shm_v5<T), BM, BN, BK, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, SmemLayoutC, S2RCopyAtomA, S2RCopyAtomB, R2SCopyAtomC, S2GCopyAtomC, S2GCopyC>,
                         hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    
    gemm_shm_v5<T, BM, BN, BK, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, SmemLayoutC, S2RCopyAtomA, S2RCopyAtomB, R2SCopyAtomC, S2GCopyAtomC, S2GCopyC>
               <<<grid, block, shm_size>>>(a, b, c, M, N, K);
}


template <typename T>
float testF16F16GemmMaxError(
    void (*gpuF16F16Gemm) (T *, T *, T *, int, int, int),
    int M, int N, int K) {

    size_t size_a = M * K * sizeof(T);
    size_t size_b = K * N * sizeof(T);
    size_t size_c = M * N * sizeof(T);

    T *h_a, *h_b, *d_a, *d_b;
    T *h_c, *d_c, *h_d_c;

    h_a = (T *)malloc(size_a);
    h_b = (T *)malloc(size_b);
    h_c = (T *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    h_d_c = (T *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = (T)(rand() / float(RAND_MAX));
    for (int i = 0; i < K * N; i++)
        h_b[i] = (T)(rand() / float(RAND_MAX));

    cpuF16F16Gemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

    gpuF16F16Gemm(d_a, d_b, d_c, M, N, K);

    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs((float)h_d_c[i] - (float)h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a); free(h_b); free(h_c); 
    hipFree(d_a); hipFree(d_b); hipFree(d_c); free(h_d_c);

    return max_error;
}

template <typename T>
float testF16F16GemmPerformance(
    void (*gpuF16F16Gemm) (T *, T *, T *, int, int, int),
    int M, int N, int K, int repeat) {

    size_t size_a = M * K * sizeof(T);
    size_t size_b = K * N * sizeof(T);
    size_t size_c = M * N * sizeof(T);

    T *d_a, *d_b;
    T *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++) {
        gpuF16F16Gemm(d_a, d_b, d_c, M, N, K);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return sec;
}


int main() {
    const int test_num = 64;
    int M_list[test_num];
    int N_list[test_num];
    int K_list[test_num];

    for (int i = 0; i < test_num; i++) {
        M_list[i] = (i + 1) * 256;
        N_list[i] = (i + 1) * 256;
        K_list[i] = (i + 1) * 256;
    }

    const int outer_repeat = 10, inner_repeat = 1;

    printf("\nalgo = Cute_HGEMM_V2\n");

    const int M = 256, N = 256, K = 256;
    float max_error = testF16F16GemmMaxError<T>(
        gemm_v3, M, N, K);
    printf("Max Error = %f\n", max_error);

    // double this_sec = testF16F16GemmPerformance<T>(
    //     gemm_v2, 8192, 8192, 8192, inner_repeat);
    for (int j = 0; j < test_num; j++) {
        int M = M_list[j], N = N_list[j], K = K_list[j];

        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int k = 0; k < outer_repeat; k++) {
            double this_sec = testF16F16GemmPerformance<T>(
                gemm_v3, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

        printf("M N K = %6d %6d %6d, ", M, N, K);
        printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
        printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
    }

    return 0;
}

