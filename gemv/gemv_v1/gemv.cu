#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <float.h>
#include "utils.h"

using T = cute::half_t;
using namespace cute;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))

#define OFFSETCOL(row, col, ld) ((col) * (ld) + (row))

template <typename T>
void cpuGemv(T *a, T *b, T *c, int m, int n, int k) {
    for (int j = 0; j < n; j++) {
        float psum = 0.0;
        for (int l = 0; l < k; l++) {
            psum += (float)a[l] * (float)b[j * k + l];
        }
        
        c[j] = (T)psum;
    }
}




template <typename T, typename ThrLayout, int BN>
__global__ void gemv_kernel(const T *Aptr, const T *Bptr, T *Cptr, int m, int n, int k) {
    int tid = threadIdx.x;
    int n_idx = blockIdx.x * BN + threadIdx.x;

    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(1, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(1, n), make_stride(n, Int<1>{}));

    int ix = blockIdx.x;

    Tensor gB = local_tile(B, make_tile(Int<BN>{}, k), make_coord(_, 0)); // (BN, k, num_tile_N)
    Tensor gC = local_tile(C, make_tile(Int<1>{}, Int<BN>{}), make_coord(0, _)); // (BM, BN) 

    const auto num_iters = size<2>(gC);
    for (int i = 0; i < num_iters; i++) {
        auto a = A(0, _);
        auto b = gB(tid, _, i);

        float psum = 0.0;
        for (int j = 0; j < k; j++) {
            psum += (float)a(j) * (float)b(j);
        }
        gC(0, tid, i) = (T)psum;
    }

    // if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     PRINT("A", A.shape())
    //     PRINT("gB", gB.shape())  
    //     PRINT("gC", gC.shape())    
    // }
}

template <typename T>
void gemv_v1(T *a, T *b, T *c, int m, int n, int k) {
    // Launch the kernel
    constexpr int BN = 128;
    using thr_layout = decltype(make_layout(make_shape(Int<BN>{})));

    int numBlocks = (n + BN - 1) / BN;

    gemv_kernel<T, thr_layout, BN><<<numBlocks, BN>>>(a, b, c, 1, n, k);
    
    // Check for any errors in kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
}


template <typename T>
float testF16F16GemmMaxError(
    void (*gpuF16F16Gemv) (T *, T *, T *, int, int, int),
    int M, int N, int K) {

    size_t size_a = 1 * K * sizeof(T);
    size_t size_b = N * K * sizeof(T);
    size_t size_c = 1 * N * sizeof(T);

    T *h_a, *h_b, *d_a, *d_b;
    T *h_c, *d_c, *h_d_c;

    h_a = (T *)malloc(size_a);
    h_b = (T *)malloc(size_b);
    h_c = (T *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    h_d_c = (T *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = (T)(rand() / float(RAND_MAX));
    for (int i = 0; i < N * K; i++)
        h_b[i] = (T)(rand() / float(RAND_MAX));

    // for (int ii = 0; ii < 10; ii++) {
    //     printf("a = %f, b = %f\n", (float)h_a[ii], (float)h_b[ii]);
    // }

    cpuGemv(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

    gpuF16F16Gemv(d_a, d_b, d_c, M, N, K); // TODO

    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);
    
    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs((float)h_d_c[i] - (float)h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a); free(h_b); free(h_c); 
    hipFree(d_a); hipFree(d_b); hipFree(d_c); free(h_d_c);

    return max_error;

}

template <typename T>
float testF16F16GemvPerformance(
    void (*gpuF16F16Gemv) (T *, T *, T *, int, int, int),
    int M, int N, int K, int repeat) {

    size_t size_a = M * K * sizeof(T);
    size_t size_b = K * N * sizeof(T);
    size_t size_c = M * N * sizeof(T);

    T *d_a, *d_b;
    T *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++) {
        gpuF16F16Gemv(d_a, d_b, d_c, M, N, K);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return sec;
}


int main() {

    const int test_num = 64;
    int M_list[test_num];
    int N_list[test_num];
    int K_list[test_num];

    for (int i = 0; i < test_num; i++) {
        M_list[i] = 1;
        N_list[i] = (i + 1) * 256;
        K_list[i] = (i + 1) * 256;
    }

    const int outer_repeat = 10, inner_repeat = 1;

    printf("\nalgo = Cute_HGEMV_V1\n");

    const int M = 1, N = 1024, K = 1024;
    float max_error = testF16F16GemmMaxError<T>(
        gemv_v1, M, N, K);
    printf("Max Error = %f\n", max_error);

    // for (int j = 0; j < test_num; j++) {
    //     int M = M_list[j], N = N_list[j], K = K_list[j];

    //     double max_sec = 0.0;
    //     double min_sec = DBL_MAX;
    //     double total_sec = 0.0;

    //     for (int k = 0; k < outer_repeat; k++) {
    //         double this_sec = testF16F16GemvPerformance<T>(
    //             gemv_v1, M, N, K, inner_repeat);
    //         max_sec = max(max_sec, this_sec);
    //         min_sec = min(min_sec, this_sec);
    //         total_sec += this_sec;
    //     }

    //     double avg_sec = total_sec / outer_repeat;
    //     double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

    //     printf("M N K = %6d %6d %6d, ", M, N, K);
    //     printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
    //     printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
    // }

}